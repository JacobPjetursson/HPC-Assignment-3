#include "hip/hip_runtime.h"
extern "C" {
#include "matmult.h"
#include <cblas.h>
}
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <omp.h>

// Block size when blocking for registers
#define bsx 1
#define bsy 16
// Thread block size for shared memory
#define BLOCK_SIZE 16
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

extern "C" {
    void matmult_lib(int m, int n, int k, double *A, double *B, double *C) {
        /*
        int iterations;
        if (m <= 2560)
            iterations = 500;
        else if (m <= 5120)
            iterations = 10;
        else
            iterations = 2;
        double start_time = omp_get_wtime();
        for (int i = 0; i < iterations; i++)
         */
            cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, m, n, k, 1, A, k, B, n, 0, C, n);
        //printf("%f\n", (omp_get_wtime() - start_time) / iterations);
    }
}

void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C) {
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;

    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    matmult_gpu1_kernel<<<1, 1>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

__global__
void matmult_gpu1_kernel(int m, int n, int k, double *A, double *B, double *C) {
    int i, j, l;
    for (i = 0; i < m; ++i) {
        for (l = 0; l < k; ++l) {
            for (j = 0; j < n; ++j) {
                C[i * n + j] += A[i * k + l] * B[l * n + j];
            }
        }
    }
}

void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C) {
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;
    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 16; // NUM_THREADS_IN_BLOCK
    int gridN = (int)ceil((double)n / blockSize);
    int gridM = (int)ceil((double)m / blockSize);
    dim3 dimGrid(gridN,gridM,1);
    dim3 dimBlock(blockSize, blockSize, 1);

    matmult_gpu2_kernel<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void matmult_gpu2_kernel(int m, int n, int k, double *A, double *B, double *C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col >= n || row >= m)
        return;
    int l;

    for (l = 0; l < k; ++l) {
        C[row * n + col] += A[row * k + l] * B[l * n + col];
    }

}

void matmult_gpu3(int m, int n, int k, double *A, double *B, double *C) {
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;
    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 16; // NUM_THREADS_IN_BLOCK
    int gridN = (int)ceil((double)n / blockSize);
    int gridM = (int)ceil((double)m / blockSize * 0.5);
    dim3 dimGrid(gridN,gridM,1);
    dim3 dimBlock(blockSize, blockSize, 1);

    matmult_gpu3_kernel<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void matmult_gpu3_kernel(int m, int n, int k, double *A, double *B, double *C) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    if (col >= n || row >= m)
        return;
    int l;
    if (row >= m - 1) {
        for (l = 0; l < k; ++l) {
            C[row * n + col] += A[row * k + l] * B[l * n + col];
        }
    } else {
        for (l = 0; l < k; ++l) {
            C[row * n + col] += A[row * k + l] * B[l * n + col];
            C[(row+1) * n + col] += A[(row+1) * k + l] * B[l * n + col];
        }
    }

}

void matmult_gpu4(int m, int n, int k, double *A, double *B, double *C) {
    hipSetDevice(1);
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;
    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 16; // NUM_THREADS_IN_BLOCK
    int gridN = (int)ceil((double)n / blockSize / bsx);
    int gridM = (int)ceil((double)m / blockSize / bsy);
    dim3 dimGrid(gridN,gridM,1);
    dim3 dimBlock(blockSize, blockSize, 1);

    matmult_gpu4_kernel<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void matmult_gpu4_kernel(int m, int n, int k, double *A, double *B, double *C) {
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * bsx;
    int row = (blockIdx.y * blockDim.y + threadIdx.y) * bsy;
    if (col >= n || row >= m)
        return;

    double C_reg[bsx * bsy];
    int l, i, j;
    for (i = 0; i < bsx * bsy; i++)
        C_reg[i] = 0.0;

    int loopJ = MIN(bsx, (n-col));
    int loopI = MIN(bsy, (m-row));
    for (j = 0; j < loopJ; j++) {
        for (i = 0; i < loopI; i++) {
            for (l = 0; l < k; ++l) {
                C_reg[i * bsx + j] += A[(row+i) * k + l] * B[l * n + col + j];
            }
            C[(row+i) * n + col+j] = C_reg[i * bsx + j];
        }
    }

}

void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C)
{
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;
    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(n / dimBlock.x, m / dimBlock.y);
    matmult_gpu5_kernel<<<dimGrid, dimBlock>>>(m, n, k, d_A, d_B, d_C);

    hipDeviceSynchronize();
    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void matmult_gpu5_kernel(int m, int n, int k, double *A, double *B, double *C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    double *Csub = &C[n * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];
    double Cvalue = 0;

    int row = threadIdx.y;
    int col = threadIdx.x;

    for (int l = 0; l < (k / BLOCK_SIZE); ++l) {
        double *Asub = &A[k * BLOCK_SIZE * blockRow + BLOCK_SIZE * l];
        double *Bsub = &B[n * BLOCK_SIZE * l + BLOCK_SIZE * blockCol];

        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[row][col] = Asub[row * k + col];
        Bs[row][col] = Bsub[row * n + col];

        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; ++i)
            Cvalue += As[row][i] * Bs[i][col];
        __syncthreads();
    }

    Csub[row * m + col] = Cvalue;
}

void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C)
{
    double *d_A, *d_B, *d_C;
    for(int i = 0; i < m * n; i++)
        C[i] = 0;
    hipMalloc( (void **)&d_A, m * k * sizeof(double));
    hipMalloc( (void **)&d_B, n * k * sizeof(double));
    hipMalloc( (void **)&d_C, m * n * sizeof(double));
    hipMemcpy(d_A, A, m * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, n * k * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(double), hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    hipMemcpy(C, d_C, m * n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}
